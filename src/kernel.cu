#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include "main.hpp"

// ---------------------------------------------------------------------------------------------- LAUNCHERS ------------------------------------------------------------------------------------------------------------

 __global__ void calculateHydrophobicBondKernel(float* posA_x, float* posA_y, float* posA_z,
                                           float* posB_x, float* posB_y, float* posB_z,
                                           float* distances, int numA, int numB) {
    // Calcola gli indici bidimensionali del thread all'interno della griglia
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per posA (molecola A)
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per posB (molecola B)
    float distance;

    // Verifica che gli indici siano validi
    if (i < numA && j < numB) {
        // Calcolo della distanza euclidea tra l'atomo i di molA e l'atomo j di molB
        float dx = posA_x[i] - posB_x[j];
        float dy = posA_y[i] - posB_y[j];
        float dz = posA_z[i] - posB_z[j];
        distance = sqrtf(dx * dx + dy * dy + dz * dz);
        
        if(distance <= DISTANCE_HYDROPHOBIC) { // Salva i risultati solo se rispettano i requisiti
            distances[i * numB + j] = distance;
        } else{
            distances[i * numB + j] = -1.0f;  // Nessuna interazione
        }
    }
}


__global__ void calculateHydrogenBondKernel(float* donor_x, float* donor_y, float* donor_z,
                                            float* hydrogen_x, float* hydrogen_y, float* hydrogen_z,
                                            float* acceptor_x, float* acceptor_y, float* acceptor_z,
                                            float* distances, float* angles, int numDonors, int numAcceptors) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per il donatore
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per l'accettore

    if (i < numDonors && j < numAcceptors) {
        // Calcolo della distanza euclidea tra donatore e accettore
        float dx = donor_x[i] - acceptor_x[j];
        float dy = donor_y[i] - acceptor_y[j];
        float dz = donor_z[i] - acceptor_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        // Calcolo dell'angolo tra donatore, idrogeno e accettore
        float hx = hydrogen_x[i], hy = hydrogen_y[i], hz = hydrogen_z[i];
        float dhx = donor_x[i] - hx, dhy = donor_y[i] - hy, dhz = donor_z[i] - hz;
        float ahx = acceptor_x[j] - hx, ahy = acceptor_y[j] - hy, ahz = acceptor_z[j] - hz;

        float dotProduct = dhx * ahx + dhy * ahy + dhz * ahz;
        float mag_dh = sqrtf(dhx * dhx + dhy * dhy + dhz * dhz);
        float mag_ah = sqrtf(ahx * ahx + ahy * ahy + ahz * ahz);
        float angle = acosf(dotProduct / (mag_dh * mag_ah)) * 180.0f / M_PI;

        // Salva le distanze e gli angoli solo se soddisfano i criteri
        if (distance <= DISTANCE_HYDROGENBOND && angle >= MIN_ANGLE_HYDROGENBOND && angle <= MAX_ANGLE_HYDROGENBOND) {
            distances[i * numAcceptors + j] = distance;
            angles[i * numAcceptors + j] = angle;
        } else {
            distances[i * numAcceptors + j] = -1.0f;  // Usa un valore negativo per indicare nessuna interazione
        }
    }
}

__global__ void calculateHalogenBondKernel(float* donor_x, float* donor_y, float* donor_z,
                                           float* halogen_x, float* halogen_y, float* halogen_z,
                                           float* acceptor_x, float* acceptor_y, float* acceptor_z,
                                           float* any_x, float* any_y, float* any_z,
                                           float* distances, float* firstAngles, float* secondAngles,
                                           int numDonors, int numAcceptors) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per i donatori
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per gli accettori

    if (i < numDonors && j < numAcceptors) {
        // Calcolo della distanza euclidea tra il donatore e l'accettore
        float dx = donor_x[i] - acceptor_x[j];
        float dy = donor_y[i] - acceptor_y[j];
        float dz = donor_z[i] - acceptor_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        // Calcolo degli angoli
        float hx = halogen_x[i], hy = halogen_y[i], hz = halogen_z[i];
        float ax = acceptor_x[j], ay = acceptor_y[j], az = acceptor_z[j];
        float anyx = any_x[j], anyy = any_y[j], anyz = any_z[j];

        // Primo angolo: tra donatore, alogeno e accettore
        float dhx = donor_x[i] - hx, dhy = donor_y[i] - hy, dhz = donor_z[i] - hz;
        float ahx = ax - hx, ahy = ay - hy, ahz = az - hz;
        float dotProduct1 = dhx * ahx + dhy * ahy + dhz * ahz;
        float mag_dh = sqrtf(dhx * dhx + dhy * dhy + dhz * dhz);
        float mag_ah = sqrtf(ahx * ahx + ahy * ahy + ahz * ahz);
        float firstAngle = acosf(dotProduct1 / (mag_dh * mag_ah)) * 180.0f / M_PI;

        // Secondo angolo: tra accettore, alogeno e "any"
        float ahhx = ax - hx, ahhy = ay - hy, ahhz = az - hz;
        float aax = anyx - ax, aay = anyy - ay, aaz = anyz - az;
        float dotProduct2 = ahhx * aax + ahhy * aay + ahhz * aaz;
        float mag_ahh = sqrtf(ahhx * ahhx + ahhy * ahhy + ahhz * ahhz);
        float mag_aa = sqrtf(aax * aax + aay * aay + aaz * aaz);
        float secondAngle = acosf(dotProduct2 / (mag_ahh * mag_aa)) * 180.0f / M_PI;

        // Salva le distanze e gli angoli solo se soddisfano i criteri
        if (distance <= DISTANCE_HALOGENBOND && firstAngle >= MIN_ANGLE1_HALOGENBOND && firstAngle <= MAX_ANGLE1_HALOGENBOND && 
            secondAngle >= MIN_ANGLE2_HALOGENBOND && secondAngle <= MAX_ANGLE2_HALOGENBOND) {
            distances[i * numAcceptors + j] = distance;
            firstAngles[i * numAcceptors + j] = firstAngle;
            secondAngles[i * numAcceptors + j] = secondAngle;
        } else {
            distances[i * numAcceptors + j] = -1.0f;  // Usa un valore negativo per indicare nessuna interazione
        }
    }
}

__global__ void calculateCationAnionKernel(float* cation_x, float* cation_y, float* cation_z,
                                           float* anion_x, float* anion_y, float* anion_z,
                                           float* distances, int numCations, int numAnions) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per i cationi
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per gli anioni

    if (i < numCations && j < numAnions) {
        // Calcolo della distanza tra catione e anione
        float dx = cation_x[i] - anion_x[j];
        float dy = cation_y[i] - anion_y[j];
        float dz = cation_z[i] - anion_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        // Verifica se la distanza è inferiore alla distanza massima per l'interazione ionica
        if (distance <= DISTANCE_IONIC) {
            distances[i * numAnions + j] = distance;
        } else {
            distances[i * numAnions + j] = -1.0f;  // Nessuna interazione
        }
    }
}

__global__ void calculateCationRingKernel(float* cation_x, float* cation_y, float* cation_z,
                                          float* ring_centroid_x, float* ring_centroid_y, float* ring_centroid_z,
                                          float* ring_normal_x, float* ring_normal_y, float* ring_normal_z,
                                          float* distances, float* angles, int numCations, int numRings) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per i cationi
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per gli anelli aromatici

    if (i < numCations && j < numRings) {
        // Calcolo della distanza tra catione e il centro dell'anello aromatico
        float dx = cation_x[i] - ring_centroid_x[j];
        float dy = cation_y[i] - ring_centroid_y[j];
        float dz = cation_z[i] - ring_centroid_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        // Verifica se la distanza è inferiore alla distanza massima
        if (distance <= DISTANCE_IONIC) {
            // Calcolo dell'angolo tra il catione e il vettore normale all'anello
            float dotProduct = dx * ring_normal_x[j] + dy * ring_normal_y[j] + dz * ring_normal_z[j];
            float magnitude_cation = sqrtf(dx * dx + dy * dy + dz * dz);
            float magnitude_normal = sqrtf(ring_normal_x[j] * ring_normal_x[j] +
                                           ring_normal_y[j] * ring_normal_y[j] +
                                           ring_normal_z[j] * ring_normal_z[j]);
            float angle = acosf(dotProduct / (magnitude_cation * magnitude_normal)) * 180.0f / M_PI;

            // Verifica se l'angolo è compreso nell'intervallo richiesto
            if (!(angle >= MIN_ANGLE_IONIC && angle <= MAX_ANGLE_IONIC) || angle == MIN_ANGLE_IONIC || angle == MAX_ANGLE_IONIC) {
                distances[i * numRings + j] = distance;
                angles[i * numRings + j] = angle;
            } else {
                distances[i * numRings + j] = -1.0f;  // Nessuna interazione
            }
        } else {
            distances[i * numRings + j] = -1.0f;  // Nessuna interazione
        }
    }
}

__global__ void calculatePiStackingKernel(
    const float* __restrict__ centroidA_x, const float* __restrict__ centroidA_y, const float* __restrict__ centroidA_z,
    const float* __restrict__ normalA_x,   const float* __restrict__ normalA_y,   const float* __restrict__ normalA_z,
    const float* __restrict__ centroidB_x, const float* __restrict__ centroidB_y, const float* __restrict__ centroidB_z,
    const float* __restrict__ normalB_x,   const float* __restrict__ normalB_y,   const float* __restrict__ normalB_z,
    float* __restrict__ distances,
    float* __restrict__ planesAngles,
    float* __restrict__ normalCentroidAnglesA,
    float* __restrict__ normalCentroidAnglesB,
    int numA, int numB)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x; // ring A
    const int j = blockIdx.y * blockDim.y + threadIdx.y; // ring B
    if (i >= numA || j >= numB) return;

    // Centroidi A,B
    const float cax = centroidA_x[i], cay = centroidA_y[i], caz = centroidA_z[i];
    const float cbx = centroidB_x[j], cby = centroidB_y[j], cbz = centroidB_z[j];

    // Vettore A→B
    const float vx = cbx - cax;
    const float vy = cby - cay;
    const float vz = cbz - caz;
    const float vmag = sqrtf(vx*vx + vy*vy + vz*vz) + 1e-20f;   // evita div/0

    // Normali normalizzate (se già normalizzate lato host, è ok comunque)
    float nax = normalA_x[i], nay = normalA_y[i], naz = normalA_z[i];
    float nbx = normalB_x[j], nby = normalB_y[j], nbz = normalB_z[j];

    const float nAmag = sqrtf(nax*nax + nay*nay + naz*naz) + 1e-20f;
    const float nBmag = sqrtf(nbx*nbx + nby*nby + nbz*nbz) + 1e-20f;
    nax /= nAmag; nay /= nAmag; naz /= nAmag;
    nbx /= nBmag; nby /= nBmag; nbz /= nBmag;

    // Distanza centroidi
    const float dist = vmag;

    // Angolo tra le normali ai piani (usa |cos| → [0,90]°, come CPU con abs)
    float cos_nn = nax*nbx + nay*nby + naz*nbz;
    cos_nn = fminf(1.f, fmaxf(-1.f, cos_nn));
    const float anglePlanes = acosf(fabsf(cos_nn)) * 180.0f / M_PI;

    // Angoli normale↔vettore tra centroidi (usa |cos| per simmetria con CPU)
    const float inv_vmag = 1.0f / vmag;
    float cos_a = (nax*vx + nay*vy + naz*vz) * inv_vmag;              // normale A ↔ (A→B)
    float cos_b = (nbx*(-vx) + nby*(-vy) + nbz*(-vz)) * inv_vmag;     // normale B ↔ (B→A)
    cos_a = fminf(1.f, fmaxf(-1.f, cos_a));
    cos_b = fminf(1.f, fmaxf(-1.f, cos_b));
    const float angleA = acosf(fabsf(cos_a)) * 180.0f / M_PI;
    const float angleB = acosf(fabsf(cos_b)) * 180.0f / M_PI;

    const int idx = i * numB + j;
    distances[idx]              = dist;
    planesAngles[idx]           = anglePlanes;
    normalCentroidAnglesA[idx]  = angleA;
    normalCentroidAnglesB[idx]  = angleB;
}


__global__ void calculateMetalBondKernel(float* posA_x, float* posA_y, float* posA_z,
                                           float* posB_x, float* posB_y, float* posB_z,
                                           float* distances, int numA, int numB) {
    // Calcola gli indici bidimensionali del thread all'interno della griglia
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Indice per posA (molecola A)
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // Indice per posB (molecola B)
    float distance;

    // Verifica che gli indici siano validi
    if (i < numA && j < numB) {
        // Calcolo della distanza euclidea tra l'atomo i di molA e l'atomo j di molB
        float dx = posA_x[i] - posB_x[j];
        float dy = posA_y[i] - posB_y[j];
        float dz = posA_z[i] - posB_z[j];
        distance = sqrtf(dx * dx + dy * dy + dz * dz);
        
        if(distance <= DISTANCE_METAL) { // Salva i risultati solo se rispettano i requisiti
            distances[i * numB + j] = distance;
        } else{
            distances[i * numB + j] = -1.0f;  // Nessuna interazione
        }
    }
}

// ---------------------------------------------------------------------------------------------- LAUNCHERS ------------------------------------------------------------------------------------------------------------


// Funzione wrapper per chiamare il kernel CUDA bidimensionale
extern "C" void launchHydrophobicBondKernel(float* d_posA_x, float* d_posA_y, float* d_posA_z,
                                       float* d_posB_x, float* d_posB_y, float* d_posB_z,
                                       float* d_distances, int numA, int numB, int blockSizeX, int blockSizeY, hipStream_t stream) {
    // Definisci la dimensione del blocco e della griglia
    dim3 threadsPerBlock(blockSizeX, blockSizeY);  // Blocchi 2D di thread
    dim3 blocksPerGrid((numA + blockSizeX - 1) / blockSizeX, 
                       (numB + blockSizeY - 1) / blockSizeY);  // Griglia 2D di blocchi

    // Lancia il kernel CUDA bidimensionale nel stream specificato
    calculateHydrophobicBondKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_posA_x, d_posA_y, d_posA_z,
                                                                             d_posB_x, d_posB_y, d_posB_z,
                                                                             d_distances, numA, numB);
}



// Funzione wrapper per chiamare il kernel CUDA per il calcolo dei legami a idrogeno
extern "C" void launchHydrogenBondKernel(float* d_donor_x, float* d_donor_y, float* d_donor_z,
                                        float* d_hydrogen_x, float* d_hydrogen_y, float* d_hydrogen_z,
                                        float* d_acceptor_x, float* d_acceptor_y, float* d_acceptor_z,
                                        float* d_distances, float* d_angles,
                                        int numDonors, int numAcceptors, int blockSizeX, int blockSizeY) {
    // Definisci la dimensione del blocco e della griglia
    dim3 threadsPerBlock(blockSizeX, blockSizeY);  // Blocchi 2D di thread
    dim3 blocksPerGrid((numDonors + blockSizeX - 1) / blockSizeX, 
                        (numAcceptors + blockSizeY - 1) / blockSizeY);  // Griglia 2D di blocchi

    // Lancia il kernel CUDA bidimensionale per il calcolo dei legami a idrogeno
    calculateHydrogenBondKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_donor_x, d_donor_y, d_donor_z,
        d_hydrogen_x, d_hydrogen_y, d_hydrogen_z,
        d_acceptor_x, d_acceptor_y, d_acceptor_z,
        d_distances, d_angles,
        numDonors, numAcceptors);
}

extern "C" void launchHalogenBondKernel(float* d_donor_x, float* d_donor_y, float* d_donor_z,
                                        float* d_halogen_x, float* d_halogen_y, float* d_halogen_z,
                                        float* d_acceptor_x, float* d_acceptor_y, float* d_acceptor_z,
                                        float* d_any_x, float* d_any_y, float* d_any_z,
                                        float* d_distances, float* d_firstAngles, float* d_secondAngles,
                                        int numDonors, int numAcceptors, int blockSizeX, int blockSizeY, hipStream_t stream) {
    // Definisci la dimensione dei blocchi e della griglia
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    dim3 blocksPerGrid((numDonors + blockSizeX - 1) / blockSizeX, 
                       (numAcceptors + blockSizeY - 1) / blockSizeY);

    // Lancia il kernel per il calcolo dei legami di alogeni nel stream specificato
    calculateHalogenBondKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        d_donor_x, d_donor_y, d_donor_z,
        d_halogen_x, d_halogen_y, d_halogen_z,
        d_acceptor_x, d_acceptor_y, d_acceptor_z,
        d_any_x, d_any_y, d_any_z,
        d_distances, d_firstAngles, d_secondAngles,
        numDonors, numAcceptors);
}

extern "C" void launchIonicInteractionsKernel_CationAnion(float* d_cation_x, float* d_cation_y, float* d_cation_z,
                                                          float* d_anion_x, float* d_anion_y, float* d_anion_z,
                                                          float* d_distances, int numCations, int numAnions, 
                                                          int blockSizeX, int blockSizeY) {
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    dim3 blocksPerGrid((numCations + blockSizeX - 1) / blockSizeX, 
                       (numAnions + blockSizeY - 1) / blockSizeY);

    // Lancia il kernel per Cationi-Anioni
    calculateCationAnionKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_cation_x, d_cation_y, d_cation_z,
        d_anion_x, d_anion_y, d_anion_z,
        d_distances, numCations, numAnions);
}

extern "C" void launchIonicInteractionsKernel_CationRing(float* d_cation_x, float* d_cation_y, float* d_cation_z,
                                                         float* d_ring_centroid_x, float* d_ring_centroid_y, float* d_ring_centroid_z,
                                                         float* d_ring_normal_x, float* d_ring_normal_y, float* d_ring_normal_z,
                                                         float* d_distances, float* d_angles, int numCations, int numRings, 
                                                         int blockSizeX, int blockSizeY) {
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    dim3 blocksPerGrid((numCations + blockSizeX - 1) / blockSizeX, 
                       (numRings + blockSizeY - 1) / blockSizeY);

    // Lancia il kernel per Cationi-Anelli Aromatici
    calculateCationRingKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_cation_x, d_cation_y, d_cation_z,
        d_ring_centroid_x, d_ring_centroid_y, d_ring_centroid_z,
        d_ring_normal_x, d_ring_normal_y, d_ring_normal_z,
        d_distances, d_angles, numCations, numRings);
}

extern "C" void launchPiStackingKernel(float* d_centroidA_x, float* d_centroidA_y, float* d_centroidA_z,
                            float* d_normalA_x,   float* d_normalA_y,   float* d_normalA_z,
                            float* d_centroidB_x, float* d_centroidB_y, float* d_centroidB_z,
                            float* d_normalB_x,   float* d_normalB_y,   float* d_normalB_z,
                            float* d_distances, float* d_planesAngles,
                            float* d_normalCentroidAnglesA, float* d_normalCentroidAnglesB,
                            int numRingsA, int numRingsB, int blockSizeX, int blockSizeY)
{
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    dim3 blocksPerGrid((numRingsA + blockSizeX - 1) / blockSizeX,
                       (numRingsB + blockSizeY - 1) / blockSizeY);

    calculatePiStackingKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_centroidA_x, d_centroidA_y, d_centroidA_z,
        d_normalA_x,   d_normalA_y,   d_normalA_z,
        d_centroidB_x, d_centroidB_y, d_centroidB_z,
        d_normalB_x,   d_normalB_y,   d_normalB_z,
        d_distances, d_planesAngles, d_normalCentroidAnglesA, d_normalCentroidAnglesB,
        numRingsA, numRingsB
    );
}

extern "C" void launchMetalBondKernel(float* d_posA_x, float* d_posA_y, float* d_posA_z,
                                       float* d_posB_x, float* d_posB_y, float* d_posB_z,
                                       float* d_distances, int numA, int numB, int blockSizeX, int blockSizeY, hipStream_t stream) {
    // Definisci la dimensione del blocco e della griglia
    dim3 threadsPerBlock(blockSizeX, blockSizeY);  // Blocchi 2D di thread
    dim3 blocksPerGrid((numA + blockSizeX - 1) / blockSizeX, 
                       (numB + blockSizeY - 1) / blockSizeY);  // Griglia 2D di blocchi

    // Lancia il kernel CUDA bidimensionale nel stream specificato
    calculateMetalBondKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_posA_x, d_posA_y, d_posA_z,
                                                                             d_posB_x, d_posB_y, d_posB_z,
                                                                             d_distances, numA, numB);
}






