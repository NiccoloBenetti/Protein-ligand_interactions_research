#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include "main.hpp"

// ---------------------------------------------------------------------------------------------- KERNELS ------------------------------------------------------------------------------------------------------------

__global__ void calculateHydrophobicBondKernel(float* posA_x, float* posA_y, float* posA_z,
                                               float* posB_x, float* posB_y, float* posB_z,
                                               float* distances, int numA, int numB)
{
    // Mappatura coalescente: B -> X, A -> Y
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // indice su B
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // indice su A

    if (i < numA && j < numB) {
        float dx = posA_x[i] - posB_x[j];
        float dy = posA_y[i] - posB_y[j];
        float dz = posA_z[i] - posB_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        distances[i * numB + j] = (distance <= DISTANCE_HYDROPHOBIC) ? distance : -1.0f;
    }
}

// --- DOPO (COERENTE: B→X=j, A→Y=i) ---
#ifndef HBOND_MIN_COS
// 130° di default; per 120° usa -DHBOND_MIN_COS=-0.5f
#define HBOND_MIN_COS -0.64278764f
#endif

#ifndef HBOND_RELAX_IF_NO_H
// metti -DHBOND_RELAX_IF_NO_H=1 se vuoi il fallback "solo distanza"
#define HBOND_RELAX_IF_NO_H 0
#endif

__global__ void calculateHydrogenBondKernel(const float* __restrict__ donor_x,
                                            const float* __restrict__ donor_y,
                                            const float* __restrict__ donor_z,
                                            const float* __restrict__ hydrogen_x,
                                            const float* __restrict__ hydrogen_y,
                                            const float* __restrict__ hydrogen_z,
                                            const float* __restrict__ acceptor_x,
                                            const float* __restrict__ acceptor_y,
                                            const float* __restrict__ acceptor_z,
                                            float* __restrict__ distances,
                                            int numDonors, int numAcceptors)
{
    // B -> X (acceptors), A -> Y (donors)  **come negli altri kernel**
    const int j = blockIdx.x * blockDim.x + threadIdx.x;  // accettore
    const int i = blockIdx.y * blockDim.y + threadIdx.y;  // donatore
    if (i >= numDonors || j >= numAcceptors) return;

    const int idx = i * numAcceptors + j;

    // ---- distanza^2 e early-out ----
    const float dx = donor_x[i] - acceptor_x[j];
    const float dy = donor_y[i] - acceptor_y[j];
    const float dz = donor_z[i] - acceptor_z[j];
    const float d2 = dx*dx + dy*dy + dz*dz;
    const float thr2 = (float)(DISTANCE_HYDROGENBOND * DISTANCE_HYDROGENBOND);
    if (d2 > thr2) { distances[idx] = -1.0f; return; }

    // ---- vettori per l’angolo D–H…H–A ----
    const float hx = hydrogen_x[i], hy = hydrogen_y[i], hz = hydrogen_z[i];
    float dhx = donor_x[i]    - hx, dhy = donor_y[i]    - hy, dhz = donor_z[i]    - hz;
    float ahx = acceptor_x[j] - hx, ahy = acceptor_y[j] - hy, ahz = acceptor_z[j] - hz;

    // Fallback opzionale se l’H non è disponibile/valido (p.es. acque senza H espliciti)
    const float dh2 = dhx*dhx + dhy*dhy + dhz*dhz;
#if HBOND_RELAX_IF_NO_H
    if (dh2 < 1e-12f) { distances[idx] = sqrtf(d2); return; }
#else
    if (dh2 < 1e-12f) { distances[idx] = -1.0f; return; }
#endif

    const float ah2 = ahx*ahx + ahy*ahy + ahz*ahz;
    const float inv_dh = rsqrtf(dh2 + 1e-20f);
    const float inv_ah = rsqrtf(ah2 + 1e-20f);
    float cosang = (dhx*ahx + dhy*ahy + dhz*ahz) * (inv_dh * inv_ah);
    cosang = fminf(1.f, fmaxf(-1.f, cosang));

    // Angolo minimo: cosθ ≤ HBOND_MIN_COS (130° default)
    if (cosang > HBOND_MIN_COS) { distances[idx] = -1.0f; return; }

    distances[idx] = sqrtf(d2);
}


__global__ void calculateHalogenBondKernel(float* donor_x, float* donor_y, float* donor_z,
                                           float* halogen_x, float* halogen_y, float* halogen_z,
                                           float* acceptor_x, float* acceptor_y, float* acceptor_z,
                                           float* any_x, float* any_y, float* any_z,
                                           float* distances,int numDonors, int numAcceptors)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;  // donatore
    const int j = blockIdx.y * blockDim.y + threadIdx.y;  // accettore
    if (i >= numDonors || j >= numAcceptors) return;

    const int idx = i * numAcceptors + j;

    // ---- distanza^2 e early-out ----
    const float dx = donor_x[i] - acceptor_x[j];
    const float dy = donor_y[i] - acceptor_y[j];
    const float dz = donor_z[i] - acceptor_z[j];
    const float d2 = dx*dx + dy*dy + dz*dz;
    const float thr2 = (float)(DISTANCE_HALOGENBOND * DISTANCE_HALOGENBOND);
    if (d2 > thr2) { distances[idx] = -1.0f; return; }

    // Vettori per l’angolo 1: D–Hal … Hal–A
    const float hx = halogen_x[i], hy = halogen_y[i], hz = halogen_z[i];
    const float ax = acceptor_x[j], ay = acceptor_y[j], az = acceptor_z[j];
    const float anyx = any_x[j], anyy = any_y[j], anyz = any_z[j];

    const float dhx = donor_x[i]-hx,  dhy = donor_y[i]-hy,  dhz = donor_z[i]-hz;
    const float ahx = ax - hx,        ahy = ay - hy,        ahz = az - hz;

    float inv_dh = rsqrtf(dhx*dhx + dhy*dhy + dhz*dhz + 1e-20f);
    float inv_ah = rsqrtf(ahx*ahx + ahy*ahy + ahz*ahz + 1e-20f);
    float cos1 = (dhx*ahx + dhy*ahy + dhz*ahz) * (inv_dh * inv_ah);
    cos1 = fminf(1.f, fmaxf(-1.f, cos1));

    // 130–180° => cosθ ≤ cos(130°)
    if (cos1 > -0.64278764f) { distances[idx] = -1.0f; return; }

    // Vettori per l’angolo 2: Hal–A … A–Any
    const float ahhx = ax - hx, ahhy = ay - hy, ahhz = az - hz;
    const float aax  = anyx - ax, aay  = anyy - ay, aaz  = anyz - az;

    float inv_ahh = rsqrtf(ahhx*ahhx + ahhy*ahhy + ahhz*ahhz + 1e-20f);
    float inv_aa  = rsqrtf(aax*aax   + aay*aay   + aaz*aaz   + 1e-20f);
    float cos2 = (ahhx*aax + ahhy*aay + ahhz*aaz) * (inv_ahh * inv_aa);
    cos2 = fminf(1.f, fmaxf(-1.f, cos2));

    // 80–140° => cos(140°) ≤ cosθ ≤ cos(80°)
    if (!(cos2 >= -0.76604444f && cos2 <= 0.17364818f)) { distances[idx] = -1.0f; return; }

    // ---- sqrt/acos solo per i validi ----
    distances[idx] = sqrtf(d2);
}


__global__ void calculateCationAnionKernel(float* cation_x, float* cation_y, float* cation_z,
                                           float* anion_x,  float* anion_y,  float* anion_z,
                                           float* distances, int numCations, int numAnions)
{
    // B -> X (anions) ; A -> Y (cations)
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // anione
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // catione

    if (i < numCations && j < numAnions) {
        float dx = cation_x[i] - anion_x[j];
        float dy = cation_y[i] - anion_y[j];
        float dz = cation_z[i] - anion_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        distances[i * numAnions + j] = (distance <= DISTANCE_IONIC) ? distance : -1.0f;
    }
}

__global__ void calculateCationRingKernel(float* cation_x, float* cation_y, float* cation_z,
                                          float* ring_centroid_x, float* ring_centroid_y, float* ring_centroid_z,
                                          float* ring_normal_x,   float* ring_normal_y,   float* ring_normal_z,
                                          float* distances, float* angles, int numCations, int numRings)
{
    // B -> X (rings) ; A -> Y (cations)
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // anello
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // catione

    if (i < numCations && j < numRings) {
        float dx = cation_x[i] - ring_centroid_x[j];
        float dy = cation_y[i] - ring_centroid_y[j];
        float dz = cation_z[i] - ring_centroid_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        if (distance <= DISTANCE_IONIC) {
            float dotProduct = dx * ring_normal_x[j] + dy * ring_normal_y[j] + dz * ring_normal_z[j];
            float magnitude_cation = sqrtf(dx * dx + dy * dy + dz * dz);
            float magnitude_normal = sqrtf(ring_normal_x[j] * ring_normal_x[j] +
                                           ring_normal_y[j] * ring_normal_y[j] +
                                           ring_normal_z[j] * ring_normal_z[j]);
            float angle = acosf(dotProduct / (magnitude_cation * magnitude_normal)) * 180.0f / M_PI;

            if (!(angle >= MIN_ANGLE_IONIC && angle <= MAX_ANGLE_IONIC) ||
                angle == MIN_ANGLE_IONIC || angle == MAX_ANGLE_IONIC)
            {
                distances[i * numRings + j] = distance;
                angles[i * numRings + j]    = angle;
            } else {
                distances[i * numRings + j] = -1.0f;
            }
        } else {
            distances[i * numRings + j] = -1.0f;
        }
    }
}

__global__ void calculatePiStackingKernel(
    const float* __restrict__ centroidA_x, const float* __restrict__ centroidA_y, const float* __restrict__ centroidA_z,
    const float* __restrict__ normalA_x,   const float* __restrict__ normalA_y,   const float* __restrict__ normalA_z,
    const float* __restrict__ centroidB_x, const float* __restrict__ centroidB_y, const float* __restrict__ centroidB_z,
    const float* __restrict__ normalB_x,   const float* __restrict__ normalB_y,   const float* __restrict__ normalB_z,
    float* __restrict__ distances,
    float* __restrict__ planesAngles,
    float* __restrict__ normalCentroidAnglesA,
    float* __restrict__ normalCentroidAnglesB,
    int numA, int numB)
{
    // B -> X (j), A -> Y (i)  **COERENTE CON TUTTO IL PROGETTO**
    const int j = blockIdx.x * blockDim.x + threadIdx.x; // ring B
    const int i = blockIdx.y * blockDim.y + threadIdx.y; // ring A
    if (i >= numA || j >= numB) return;

    const int idx = i * numB + j;

    auto poison = [&](void){
        distances[idx]              = -1.0f;
        planesAngles[idx]           = 999.0f;
        normalCentroidAnglesA[idx]  = 999.0f;
        normalCentroidAnglesB[idx]  = 999.0f;
    };

    // ---- distanza^2 e early-out con soglia T-shape ----
    const float cax = centroidA_x[i], cay = centroidA_y[i], caz = centroidA_z[i];
    const float cbx = centroidB_x[j], cby = centroidB_y[j], cbz = centroidB_z[j];
    const float vx = cbx - cax, vy = cby - cay, vz = cbz - caz;
    const float v2 = vx*vx + vy*vy + vz*vz;
    const float thr2 = (float)(DISTANCE_TSHAPE * DISTANCE_TSHAPE);
    if (v2 > thr2) { poison(); return; }

    // ---- normali normalizzate ----
    float nax = normalA_x[i], nay = normalA_y[i], naz = normalA_z[i];
    float nbx = normalB_x[j], nby = normalB_y[j], nbz = normalB_z[j];
    const float inv_nA = rsqrtf(nax*nax + nay*nay + naz*naz + 1e-20f);
    const float inv_nB = rsqrtf(nbx*nbx + nby*nby + nbz*nbz + 1e-20f);
    nax *= inv_nA; nay *= inv_nA; naz *= inv_nA;
    nbx *= inv_nB; nby *= inv_nB; nbz *= inv_nB;

    // cos(piano–piano), usiamo |cos|
    float cos_nn = nax*nbx + nay*nby + naz*nbz;
    cos_nn = fminf(1.f, fmaxf(-1.f, cos_nn));
    const float abs_cnn = fabsf(cos_nn);

    // Prefiltro unione finestre: [0–30°] U [50–90°]
    // => |cos| ≥ cos30° (~0.8660)  OR  |cos| ≤ cos50° (~0.6428)
    if (!(abs_cnn >= 0.86602540f || abs_cnn <= 0.64278764f)) { poison(); return; }

    // cos(normale–centroide) per A→B e B→A, con |cos|
    const float inv_v = rsqrtf(v2 + 1e-20f);
    float cos_a = (nax*vx + nay*vy + naz*vz) * inv_v;
    float cos_b = (nbx*(-vx) + nby*(-vy) + nbz*(-vz)) * inv_v;
    cos_a = fminf(1.f, fmaxf(-1.f, cos_a));
    cos_b = fminf(1.f, fmaxf(-1.f, cos_b));

    // Prefiltro 0–33° ⇒ |cos| ≥ cos33° (~0.8387) per entrambi
    if (fabsf(cos_a) < 0.83867057f || fabsf(cos_b) < 0.83867057f) { poison(); return; }

    // ---- solo ora sqrt/acos (per scrivere valori "puliti" che l'host confronterà) ----
    const float dist = sqrtf(v2);
    distances[idx]              = dist;
    planesAngles[idx]           = acosf(fabsf(cos_nn)) * 180.0f / (float)M_PI; // [0,90]
    normalCentroidAnglesA[idx]  = acosf(fabsf(cos_a))  * 180.0f / (float)M_PI; // [0,90]
    normalCentroidAnglesB[idx]  = acosf(fabsf(cos_b))  * 180.0f / (float)M_PI; // [0,90]
}


__global__ void calculateMetalBondKernel(float* posA_x, float* posA_y, float* posA_z,
                                         float* posB_x, float* posB_y, float* posB_z,
                                         float* distances, int numA, int numB)
{
    // B -> X, A -> Y
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // indice su B
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // indice su A

    if (i < numA && j < numB) {
        float dx = posA_x[i] - posB_x[j];
        float dy = posA_y[i] - posB_y[j];
        float dz = posA_z[i] - posB_z[j];
        float distance = sqrtf(dx * dx + dy * dy + dz * dz);

        distances[i * numB + j] = (distance <= DISTANCE_METAL) ? distance : -1.0f;
    }
}

// ---------------------------------------------------------------------------------------------- LAUNCHERS ------------------------------------------------------------------------------------------------------------

extern "C" void launchHydrophobicBondKernel(float* d_posA_x, float* d_posA_y, float* d_posA_z,
                                            float* d_posB_x, float* d_posB_y, float* d_posB_z,
                                            float* d_distances, int numA, int numB,
                                            int blockSizeX, int blockSizeY, hipStream_t stream)
{
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    // Coalescente: grid.x su B, grid.y su A
    dim3 blocksPerGrid((numB + blockSizeX - 1) / blockSizeX,
                       (numA + blockSizeY - 1) / blockSizeY);

    calculateHydrophobicBondKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        d_posA_x, d_posA_y, d_posA_z,
        d_posB_x, d_posB_y, d_posB_z,
        d_distances, numA, numB);
}

extern "C" void launchHydrogenBondKernel(float* d_donor_x, float* d_donor_y, float* d_donor_z,
                                         float* d_hydrogen_x, float* d_hydrogen_y, float* d_hydrogen_z,
                                         float* d_acceptor_x, float* d_acceptor_y, float* d_acceptor_z,
                                         float* d_distances,
                                         int numDonors, int numAcceptors,
                                         int blockSizeX, int blockSizeY)
{
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    // grid.x = B(acceptors), grid.y = A(donors)
    dim3 blocksPerGrid((numAcceptors + blockSizeX - 1) / blockSizeX,
                       (numDonors    + blockSizeY - 1) / blockSizeY);

    calculateHydrogenBondKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_donor_x, d_donor_y, d_donor_z,
        d_hydrogen_x, d_hydrogen_y, d_hydrogen_z,
        d_acceptor_x, d_acceptor_y, d_acceptor_z,
        d_distances,
        numDonors, numAcceptors);
}

extern "C" void launchHalogenBondKernel(float* d_donor_x, float* d_donor_y, float* d_donor_z,
                                        float* d_halogen_x, float* d_halogen_y, float* d_halogen_z,
                                        float* d_acceptor_x, float* d_acceptor_y, float* d_acceptor_z,
                                        float* d_any_x, float* d_any_y, float* d_any_z,
                                        float* d_distances, int numDonors, int numAcceptors,
                                        int blockSizeX, int blockSizeY, hipStream_t stream)
{
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    // grid.x = B(acceptors), grid.y = A(donors)
    dim3 blocksPerGrid((numAcceptors + blockSizeX - 1) / blockSizeX,
                       (numDonors    + blockSizeY - 1) / blockSizeY);

    calculateHalogenBondKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        d_donor_x, d_donor_y, d_donor_z,
        d_halogen_x, d_halogen_y, d_halogen_z,
        d_acceptor_x, d_acceptor_y, d_acceptor_z,
        d_any_x, d_any_y, d_any_z,
        d_distances, numDonors, numAcceptors);
}

extern "C" void launchIonicInteractionsKernel_CationAnion(float* d_cation_x, float* d_cation_y, float* d_cation_z,
                                                          float* d_anion_x,  float* d_anion_y,  float* d_anion_z,
                                                          float* d_distances, int numCations, int numAnions,
                                                          int blockSizeX, int blockSizeY)
{
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    // grid.x = B(anions), grid.y = A(cations)
    dim3 blocksPerGrid((numAnions + blockSizeX - 1) / blockSizeX,
                       (numCations + blockSizeY - 1) / blockSizeY);

    calculateCationAnionKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_cation_x, d_cation_y, d_cation_z,
        d_anion_x,  d_anion_y,  d_anion_z,
        d_distances, numCations, numAnions);
}

extern "C" void launchIonicInteractionsKernel_CationRing(float* d_cation_x, float* d_cation_y, float* d_cation_z,
                                                         float* d_ring_centroid_x, float* d_ring_centroid_y, float* d_ring_centroid_z,
                                                         float* d_ring_normal_x,   float* d_ring_normal_y,   float* d_ring_normal_z,
                                                         float* d_distances, float* d_angles,
                                                         int numCations, int numRings,
                                                         int blockSizeX, int blockSizeY)
{
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    // grid.x = B(rings), grid.y = A(cations)
    dim3 blocksPerGrid((numRings   + blockSizeX - 1) / blockSizeX,
                       (numCations + blockSizeY - 1) / blockSizeY);

    calculateCationRingKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_cation_x, d_cation_y, d_cation_z,
        d_ring_centroid_x, d_ring_centroid_y, d_ring_centroid_z,
        d_ring_normal_x,   d_ring_normal_y,   d_ring_normal_z,
        d_distances, d_angles, numCations, numRings);
}

extern "C" void launchPiStackingKernel(float* d_centroidA_x, float* d_centroidA_y, float* d_centroidA_z,
                                       float* d_normalA_x,   float* d_normalA_y,   float* d_normalA_z,
                                       float* d_centroidB_x, float* d_centroidB_y, float* d_centroidB_z,
                                       float* d_normalB_x,   float* d_normalB_y,   float* d_normalB_z,
                                       float* d_distances, float* d_planesAngles,
                                       float* d_normalCentroidAnglesA, float* d_normalCentroidAnglesB,
                                       int numRingsA, int numRingsB,
                                       int blockSizeX, int blockSizeY)
{
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    // grid.x = B(ringsB), grid.y = A(ringsA)
    dim3 blocksPerGrid((numRingsB + blockSizeX - 1) / blockSizeX,
                       (numRingsA + blockSizeY - 1) / blockSizeY);

    calculatePiStackingKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_centroidA_x, d_centroidA_y, d_centroidA_z,
        d_normalA_x,   d_normalA_y,   d_normalA_z,
        d_centroidB_x, d_centroidB_y, d_centroidB_z,
        d_normalB_x,   d_normalB_y,   d_normalB_z,
        d_distances, d_planesAngles, d_normalCentroidAnglesA, d_normalCentroidAnglesB,
        numRingsA, numRingsB);
}

extern "C" void launchMetalBondKernel(float* d_posA_x, float* d_posA_y, float* d_posA_z,
                                      float* d_posB_x, float* d_posB_y, float* d_posB_z,
                                      float* d_distances, int numA, int numB,
                                      int blockSizeX, int blockSizeY, hipStream_t stream)
{
    dim3 threadsPerBlock(blockSizeX, blockSizeY);
    // grid.x = B, grid.y = A
    dim3 blocksPerGrid((numB + blockSizeX - 1) / blockSizeX,
                       (numA + blockSizeY - 1) / blockSizeY);

    calculateMetalBondKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        d_posA_x, d_posA_y, d_posA_z,
        d_posB_x, d_posB_y, d_posB_z,
        d_distances, numA, numB);
}
